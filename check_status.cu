#include <iostream>
#include <hip/hip_runtime.h>

int main() {
    int deviceCount;
    hipError_t err = hipGetDeviceCount(&deviceCount);
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
        return -1;
    }

    for (int device = 0; device < deviceCount; ++device) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, device);
        std::cout << "Device " << device << ": " << deviceProp.name << std::endl;
        std::cout << "  Compute capability: " << deviceProp.major << "." << deviceProp.minor << std::endl;
        std::cout << "  Total global memory: " << deviceProp.totalGlobalMem << " bytes" << std::endl;
    }

    return 0;
}
